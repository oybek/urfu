
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <iostream>
#include <fstream>

#include "matrix.h"

int main()
{
	// ������� �������� ��� ������� �� �����
	matrix<int> m;
	int rown, coln;

#define LOCAL_PATH "E:\\urfu\\parallel-computing\\cuda-matrix-sum\\Debug\\"

	std::ifstream fin( LOCAL_PATH "m.txt" );
	std::ofstream fout( LOCAL_PATH "m_double.txt" );
	fin >> rown >> coln;

	m.resize(rown, coln);
	for (int i = 0; i < rown; ++i)
	{
		for (int j = 0; j < coln; ++j)
			fin >> m[i][j];
	}

	matrix<int> m_sum = m + m;

	m_sum.info();

	for (int i = 0; i < rown; ++i)
	{
		for (int j = 0; j < coln; ++j)
		{
			fout << m_sum[i][j] << ' ';
		}
		fout << std::endl;
	}

    return 0;
}
